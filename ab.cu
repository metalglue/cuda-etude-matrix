#include "hip/hip_runtime.h"
/* vim: set sw=4 sts=4 sta : */

#include <cstdlib>
#include <stdio.h>
#include <assert.h>
#include <cutil.h>
#include <cutil_inline.h>

const int DIM = 1000;
const int DIM_SMALL = 200;
typedef float number;

/*        */
/* matrix */
/*        */

class matrix {
public:
    static matrix *new_random_filled(int height, int width);
    static matrix *new_garbage(int height, int width);
    static void delete_matrix(matrix *m);
    class iter_all {
    public:
        iter_all(const matrix *m) { pix = &m->items[0]; beyond = &m->items[m->height * m->width]; }
        void operator ++(int) { pix++; }
        number *operator *() { return pix == beyond ? 0 : pix; }
        void ok() { assert(pix <= beyond); }
    private:
        number *pix;
        number *beyond;
    };
    void size(int *height_, int *width_) const { *height_ = height; *width_ = width; }
    void show() const ;
    class iter_row {
    public:
        iter_row(const matrix *m, int row) { pix = &m->items[row * m->width]; beyond = &m->items[(row + 1) * m->width]; }
        void operator ++(int) { pix++; }
        void operator +=(int d) { pix += d; }
        number *operator *() { return pix == beyond ? 0 : pix; }
        void ok() { assert(pix <= beyond); }
    private:
        number *pix;
        number *beyond;
    };
    class iter_col {
    public:
        iter_col(const matrix *m, int col) : width(m->width) { pix = &m->items[col]; beyond = &m->items[m->height * m->width + col]; }
        void operator ++(int) { pix += width; }
        number *operator *() { return pix == beyond ? 0 : pix; }
        void ok() { assert(pix <= beyond); }
    private:
        const int width;
        number *pix;
        number *beyond;
    };
    static bool eq(const matrix *a, const matrix *b);
    static long mul_ijk(const matrix *a, const matrix *b, matrix *r);
    static long mul_kij(const matrix *a, const matrix *b, matrix *r);
    static long mul_ikj(const matrix *a, const matrix *b, matrix *r);
    static long mul_cuda_1(const matrix *a, const matrix *b, matrix *r);
    static long mul_cuda_2(const matrix *a, const matrix *b, matrix *r);
private:
    matrix();
    matrix(int height_, int width_) : height(height_), width(width_) {}
private:
    const int height, width;
    number *items;
};

matrix *matrix::new_garbage(int height, int width)
{
    matrix *m = new matrix(height, width);
    m->items = new number[height * width];
    return m;
}

matrix *matrix::new_random_filled(int height, int width)
{
    matrix *m = new_garbage(height, width);
    for (iter_all i(m); *i != 0; i++) {
        number *n = *i;
        *n = rand() % 10;
    }
    return m;
}

void matrix::delete_matrix(matrix *m)
{
    delete[] m->items;
    delete m;
}

void matrix::show() const
{
    int height, width;
    size(&height, &width);
    matrix::iter_all i(this);
    printf("(\n");
    for (int line = 0; line < height; line++) {
        for (int column = 0; column < width; column++) {
            printf("%2.0f ", (double)**i);
            i++;
        }
        printf("\n");
    }
    printf(")\n");
}

bool matrix::eq(const matrix *a, const matrix *b)
{
    assert(a->height == b->height && a->width == b->width);
    iter_all ia(a);
    iter_all ib(b);
    for (; *ia; ia++, ib++)
        if (**ia != **ib)
            return false;
    return true;
}

long matrix::mul_ijk(const matrix *a, const matrix *b, matrix *r)
{
    int width, height;
    a->size(&height, &width);
    for (int i = 0; i < height; i++) {
        iter_row r_i(r, i);
        for (int j = 0; j < height; j++) {
            iter_row a_i(a, i);
            iter_col b_i(b, j);
            double n = 0;
            for (int k = 0; k < width; k++) {
                n += **a_i * **b_i;
                a_i++; b_i++;
            }
            **r_i = n;
            r_i++;
        }
    }
    return (long)width * width * height * 2;
}

long matrix::mul_kij(const matrix *a, const matrix *b, matrix *r)
{
    int width, height, h_div, h_mod;
    a->size(&height, &width);
    h_div = height / 16;
    h_mod = height % 16;
    for (iter_all i(r); *i; i++)
        **i = 0;
    for (int k = 0; k < width; k++) {
        iter_col a_i(a, k);
        for (int i = 0; i < height; i++) {
            number n = **a_i; a_i++;
            iter_row b_i(b, k);
            iter_row r_i(r, i);
            for (int j = 0; j < h_div; j++) {
                (*r_i)[0] += (*b_i)[0] * n;
                (*r_i)[1] += (*b_i)[1] * n;
                (*r_i)[2] += (*b_i)[2] * n;
                (*r_i)[3] += (*b_i)[3] * n;
                (*r_i)[4] += (*b_i)[4] * n;
                (*r_i)[5] += (*b_i)[5] * n;
                (*r_i)[6] += (*b_i)[6] * n;
                (*r_i)[7] += (*b_i)[7] * n;
                (*r_i)[8] += (*b_i)[8] * n;
                (*r_i)[9] += (*b_i)[9] * n;
                (*r_i)[10] += (*b_i)[10] * n;
                (*r_i)[11] += (*b_i)[11] * n;
                (*r_i)[12] += (*b_i)[12] * n;
                (*r_i)[13] += (*b_i)[13] * n;
                (*r_i)[14] += (*b_i)[14] * n;
                (*r_i)[15] += (*b_i)[15] * n;
                r_i += 16; b_i += 16;
            }
            for (int j = 0; j < h_mod; j++) {
                **r_i += **b_i * n;
                r_i++; b_i++;
            }
        }
    }
    return (long)width * width * height * 2;
}

long matrix::mul_ikj(const matrix *a, const matrix *b, matrix *r)
{
    int width, height, h_div, h_mod;
    a->size(&height, &width);
    h_div = height / 16;
    h_mod = height % 16;
    for (int i = 0; i < height; i++) {
        for (iter_row r_i(r, i); *r_i; r_i++)
            **r_i = 0;
        iter_row a_i(a, i);
        for (int k = 0; k < width; k++) {
            number n = **a_i; a_i++;
            iter_row r_i(r, i);
            iter_row b_i(b, k);
            for (int j = 0; j < h_div; j++) {
                (*r_i)[0] += (*b_i)[0] * n;
                (*r_i)[1] += (*b_i)[1] * n;
                (*r_i)[2] += (*b_i)[2] * n;
                (*r_i)[3] += (*b_i)[3] * n;
                (*r_i)[4] += (*b_i)[4] * n;
                (*r_i)[5] += (*b_i)[5] * n;
                (*r_i)[6] += (*b_i)[6] * n;
                (*r_i)[7] += (*b_i)[7] * n;
                (*r_i)[8] += (*b_i)[8] * n;
                (*r_i)[9] += (*b_i)[9] * n;
                (*r_i)[10] += (*b_i)[10] * n;
                (*r_i)[11] += (*b_i)[11] * n;
                (*r_i)[12] += (*b_i)[12] * n;
                (*r_i)[13] += (*b_i)[13] * n;
                (*r_i)[14] += (*b_i)[14] * n;
                (*r_i)[15] += (*b_i)[15] * n;
                r_i += 16; b_i += 16;
            }
            for (int j = 0; j < h_mod; j++) {
                **r_i += **b_i * n;
                r_i++; b_i++;
            }
        }
    }
    return (long)width * width * height * 2;
}

__global__ void mul_cuda_1_kernel(number *a, number *b, number *r, int height, int width)
{
    int tx = blockIdx.x;
    int ty = blockIdx.y;
    number n = 0;
    for (int k = 0; k < width; k++) {
        n += a[ty * width + k] * b[k * height + tx];
    }
    r[ty * height + tx] = n;
}

long matrix::mul_cuda_1(const matrix *a, const matrix *b, matrix *r)
{
    int height, width;
    a->size(&height, &width);
    void *aa, *bb, *rr;
    CUDA_SAFE_CALL( hipMalloc(&aa, height * width * sizeof(number)) );
    matrix::iter_all ia(a);
    CUDA_SAFE_CALL( hipMemcpy(aa, *ia, height * width * sizeof(number), hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMalloc(&bb, height * width * sizeof(number)) );
    matrix::iter_all ib(b);
    CUDA_SAFE_CALL( hipMemcpy(bb, *ib, height * width * sizeof(number), hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMalloc(&rr, height * height * sizeof(number)) );
    dim3 grid(height, height);
    dim3 block(1, 1);
    mul_cuda_1_kernel<<<grid, block>>>((number *)aa, (number *)bb, (number *)rr, height, width);
    matrix::iter_all ir(r);
    CUDA_SAFE_CALL( hipMemcpy(*ir, rr, height * height * sizeof(number), hipMemcpyDeviceToHost) );
    hipFree(rr); hipFree(bb); hipFree(aa);
    return (long)width * width * height * 2;
}

__global__ void mul_cuda_2_kernel(number *a, number *b, number *r, int height, int width)
{
    int h_div, h_mod;
    h_div = height / 16;
    h_mod = height % 16;
    for (int i = 0; i < height; i++) {
        {
            number *r_i = &r[i * height];
            number *r_i_beyond = &r[(i + 1) * height];
            for (; r_i != r_i_beyond; ++r_i)
                *r_i = 0;
        }
        number *a_i = &a[i * width];
        for (int k = 0; k < width; k++) {
            number n = *a_i++;
            number *r_i = &r[i * height];
            number *b_i = &b[k * height];
            for (int j = 0; j < h_div; j++) {
                r_i[0] += b_i[0] * n;
                r_i[1] += b_i[1] * n;
                r_i[2] += b_i[2] * n;
                r_i[3] += b_i[3] * n;
                r_i[4] += b_i[4] * n;
                r_i[5] += b_i[5] * n;
                r_i[6] += b_i[6] * n;
                r_i[7] += b_i[7] * n;
                r_i[8] += b_i[8] * n;
                r_i[9] += b_i[9] * n;
                r_i[10] += b_i[10] * n;
                r_i[11] += b_i[11] * n;
                r_i[12] += b_i[12] * n;
                r_i[13] += b_i[13] * n;
                r_i[14] += b_i[14] * n;
                r_i[15] += b_i[15] * n;
                r_i += 16; b_i += 16;
            }
            for (int j = 0; j < h_mod; j++) {
                *r_i += *b_i * n;
                ++r_i; ++b_i;
            }
        }
    }
}

long matrix::mul_cuda_2(const matrix *a, const matrix *b, matrix *r)
{
    int height, width;
    a->size(&height, &width);
    void *aa, *bb, *rr;
    CUDA_SAFE_CALL( hipMalloc(&aa, height * width * sizeof(number)) );
    matrix::iter_all ia(a);
    CUDA_SAFE_CALL( hipMemcpy(aa, *ia, height * width * sizeof(number), hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMalloc(&bb, height * width * sizeof(number)) );
    matrix::iter_all ib(b);
    CUDA_SAFE_CALL( hipMemcpy(bb, *ib, height * width * sizeof(number), hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMalloc(&rr, height * height * sizeof(number)) );
    dim3 grid(1, 1);
    dim3 block(1, 1);
    mul_cuda_2_kernel<<<grid, block>>>((number *)aa, (number *)bb, (number *)rr, height, width);
    matrix::iter_all ir(r);
    CUDA_SAFE_CALL( hipMemcpy(*ir, rr, height * height * sizeof(number), hipMemcpyDeviceToHost) );
    hipFree(rr); hipFree(bb); hipFree(aa);
    return (long)width * width * height * 2;
}

/*           */
/* stopwatch */
/*           */

#include <time.h>
#include <stdio.h>
#include <unistd.h>
#include <sys/times.h>

class stopwatch {
public:
    void start();
    void get_lap();
    void show(long flop_count) const ;
private:
    clock_t clock_start, clock_lap;
    tms tms_start, tms_lap;
};

void stopwatch::start()
{
    clock_start = ::clock();
    ::times(&tms_start);
}

void stopwatch::get_lap()
{
    clock_lap = ::clock();
    ::times(&tms_lap);
}

void stopwatch::show(long flop_count) const
{
    long ticks = ::sysconf(_SC_CLK_TCK);
    printf("Real: %.2f s\n", (double)(clock_lap - clock_start) / CLOCKS_PER_SEC);
    printf("User: %.2f s\n", (double)(tms_lap.tms_utime - tms_start.tms_utime) / ticks);
    printf(" Sys: %.2f s\n", (double)(tms_lap.tms_stime - tms_start.tms_stime) / ticks);
    printf("%.2f GFLOPS\n", (double)flop_count / (clock_lap - clock_start) * CLOCKS_PER_SEC / 1000 / 1000 / 1000);
}

/*      */
/* main */
/*      */

void
test_0001()
{
    matrix *a = matrix::new_random_filled(2, 3);
    matrix *b = matrix::new_random_filled(3, 2);
    matrix *r = matrix::new_garbage(2, 2);
    stopwatch sw;
    sw.start();
    long flop_count = matrix::mul_cuda_1(a, b, r);
    sw.get_lap();
    a->show(); b->show(); r->show();
    sw.show(flop_count);
    matrix::delete_matrix(a);
    matrix::delete_matrix(b);
    matrix::delete_matrix(r);
}

void
test_0002()
{
    matrix *a = matrix::new_random_filled(DIM, DIM);
    matrix *b = matrix::new_random_filled(DIM, DIM);
    matrix *r1 = matrix::new_garbage(DIM, DIM);
    matrix *r2 = matrix::new_garbage(DIM, DIM);
    matrix *r3 = matrix::new_garbage(DIM, DIM);
    stopwatch sw;
    sw.start();
    long flop_count;
    flop_count = matrix::mul_kij(a, b, r1);
    sw.get_lap();
    sw.show(flop_count);
    sw.start();
    flop_count = matrix::mul_ikj(a, b, r2);
    sw.get_lap();
    sw.show(flop_count);
    sw.start();
    flop_count = matrix::mul_cuda_1(a, b, r3);
    sw.get_lap();
    sw.show(flop_count);
    assert(matrix::eq(r1, r2));
    assert(matrix::eq(r2, r3));
    matrix::delete_matrix(a);
    matrix::delete_matrix(b);
    matrix::delete_matrix(r1);
    matrix::delete_matrix(r2);
    matrix::delete_matrix(r3);
}

void
test_0003()
{
    matrix *a = matrix::new_random_filled(2, 3);
    matrix *b = matrix::new_random_filled(3, 2);
    matrix *r = matrix::new_garbage(2, 2);
    stopwatch sw;
    sw.start();
    long flop_count = matrix::mul_cuda_2(a, b, r);
    sw.get_lap();
    a->show(); b->show(); r->show();
    sw.show(flop_count);
    matrix::delete_matrix(a);
    matrix::delete_matrix(b);
    matrix::delete_matrix(r);
}

void
test_0004()
{
    matrix *a = matrix::new_random_filled(DIM_SMALL, DIM_SMALL);
    matrix *b = matrix::new_random_filled(DIM_SMALL, DIM_SMALL);
    matrix *r1 = matrix::new_garbage(DIM_SMALL, DIM_SMALL);
    matrix *r2 = matrix::new_garbage(DIM_SMALL, DIM_SMALL);
    matrix *r3 = matrix::new_garbage(DIM_SMALL, DIM_SMALL);
    matrix *r4 = matrix::new_garbage(DIM_SMALL, DIM_SMALL);
    stopwatch sw;
    sw.start();
    long flop_count;
    flop_count = matrix::mul_kij(a, b, r1);
    sw.get_lap();
    sw.show(flop_count);
    sw.start();
    flop_count = matrix::mul_ikj(a, b, r2);
    sw.get_lap();
    sw.show(flop_count);
    sw.start();
    flop_count = matrix::mul_cuda_1(a, b, r3);
    sw.get_lap();
    sw.show(flop_count);
    sw.start();
    flop_count = matrix::mul_cuda_2(a, b, r4);
    sw.get_lap();
    sw.show(flop_count);
    assert(matrix::eq(r1, r2));
    assert(matrix::eq(r2, r3));
    assert(matrix::eq(r3, r4));
    matrix::delete_matrix(a);
    matrix::delete_matrix(b);
    matrix::delete_matrix(r1);
    matrix::delete_matrix(r2);
    matrix::delete_matrix(r3);
    matrix::delete_matrix(r4);
}

int main(int argc, char *argv[])
{
    CUT_DEVICE_INIT(argc, argv);

    {
        void *dummy;
        CUDA_SAFE_CALL( hipMalloc(&dummy, 1) );
        CUDA_SAFE_CALL( hipFree(dummy) );
    }

    test_0004();

    CUT_EXIT(argc, argv);
    return 0;
}

